#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"
#include "stencil.h"
#include "amul.h"
#include "stdio.h"

extern "C" __global__ void
Elastodynamic1(float* __restrict__ dux, float* __restrict__ duy, float* __restrict__ duz, 
                 float* __restrict__ ux, float* __restrict__ uy, float* __restrict__ uz,
                 int Nx, int Ny, int Nz, float wx, float wy, float wz, 
                 float* __restrict__  C1_, float  C1_mul, float* __restrict__  C2_, float  C2_mul, 
                 float* __restrict__  C3_, float  C3_mul, uint8_t PBC) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    //Do nothing if cell position is not in mesh
    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }

    // Central cell
    int I = idx(ix, iy, iz);
    float3 u0 = make_float3(ux[I], uy[I], uz[I]);
    float3 cc = make_float3(0.0,0.0,0.0);
    
    //Neighbor cell
    int I_ = idx(ix, iy, iz);
    float3 u_ = make_float3(0.0,0.0,0.0);
    float3 cc_ =make_float3(0.0,0.0,0.0);

    float3 d_ = make_float3(0.0,0.0,0.0);

    //Set output to zero at start
    dux[I] = 0.0 ;
    duy[I] = 0.0 ;
    duz[I] = 0.0 ;

    //Check if you are in vacuum region
    if (amul(C1_, C1_mul, I)==0) {
        return;
    }

    //dxx
    d_ = make_float3(0.0,0.0,0.0);
    cc = make_float3(amul(C1_, C1_mul, I),amul(C3_, C3_mul, I),amul(C3_, C3_mul, I));
    //Right neighbor
    I_ = idx(hclampx(ix+1), iy, iz);
    u_ = make_float3(ux[I_], uy[I_], uz[I_]);
    cc_ = make_float3(amul(C1_, C1_mul, I_),amul(C3_, C3_mul, I_), amul(C3_, C3_mul, I_));
    //Harmonic mean, takes also vacuum regions into account because product will be zero
    cc_ = 2*haddiv(had(cc,cc_),(cc+cc_));
    d_ = wx*wx*had(cc_,(u_-u0));
    //Left neighbour
    I_ = idx(lclampx(ix-1), iy, iz);
    u_ = make_float3(ux[I_], uy[I_], uz[I_]);
    cc_ = make_float3(amul(C1_, C1_mul, I_),amul(C3_, C3_mul, I_), amul(C3_, C3_mul, I_));
    cc_ = 2*haddiv(had(cc,cc_),(cc+cc_));
    d_ += wx*wx*had(cc_,(u_-u0));
    
    dux[I] += d_.x ;
    duy[I] += d_.y ;
    duz[I] += d_.z ;

    //dyy
    d_ = make_float3(0.0,0.0,0.0);
    cc = make_float3(amul(C3_, C3_mul, I),amul(C1_, C1_mul, I),amul(C3_, C3_mul, I));
    //Right neighbor
    I_ = idx(ix, hclampy(iy+1), iz);
    u_ = make_float3(ux[I_], uy[I_], uz[I_]);
    cc_ = make_float3(amul(C3_, C3_mul, I),amul(C1_, C1_mul, I),amul(C3_, C3_mul, I));
    //Harmonic mean, takes also vacuum regions into account because product will be zero
    cc_ = 2*haddiv(had(cc,cc_),(cc+cc_));
    d_ = wy*wy*had(cc_,(u_-u0));
    //Left neighbour
    I_ = idx(ix, lclampy(iy-1), iz);
    u_ = make_float3(ux[I_], uy[I_], uz[I_]);
    cc_ = make_float3(amul(C3_, C3_mul, I),amul(C1_, C1_mul, I),amul(C3_, C3_mul, I));
    cc_ = 2*haddiv(had(cc,cc_),(cc+cc_));
    d_ += wy*wy*had(cc_,(u_-u0));
    
    dux[I] += d_.x ;
    duy[I] += d_.y ;
    duz[I] += d_.z ;


    // //dzz
    // d_ = make_float3(0.0,0.0,0.0);
    // cc = make_float3(amul(C3_, C3_mul, I),amul(C3_, C3_mul, I),amul(C1_, C1_mul, I));
    // //If there is a right neighbor
    // I_ = idx(ix, iy, hclampz(iz+1));
    // u_ = make_float3(ux[I_], uy[I_], uz[I_]);
    // cc_ = make_float3(amul(C3_, C3_mul, I_),amul(C3_, C3_mul, I_), amul(C1_, C1_mul, I_));
    // cc_ = 2*haddiv(had(cc,cc_),(cc+cc_));
    // d_ = wz*wz*had(cc_,(u_-u0));
    // //If there is left neighbour
    // I_ = idx(ix, iy, lclampz(iz-1));
    // u_ = make_float3(ux[I_], uy[I_], uz[I_]);
    // cc_ = make_float3(amul(C3_, C3_mul, I_),amul(C3_, C3_mul, I_), amul(C1_, C1_mul, I_));
    // cc_ = 2*haddiv(had(cc,cc_),(cc+cc_));
    // d_ = wz*wz*had(cc_,(u_-u0));
    
    // dux[I] += d_.x ;
    // duy[I] += d_.y ;
    // duz[I] += d_.z ;


    // Output should be equal to:
    // dux[I] = dxx.x + dxy.y + dxz.z + dyy.x + dyx.y + dzz.x + dzx.z;
    // duy[I] = dyy.y + dyx.x + dyz.z + dxx.y + dxy.x + dzz.y + dzy.z;
    // duz[I] = dzz.z + dzx.x + dzy.y + dxx.z + dxz.x + dyy.z + dyz.y; 
}
