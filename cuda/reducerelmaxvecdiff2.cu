#include "hip/hip_runtime.h"
#include "reduce.h"
#include "atomicf.h"
#include "float3.h"

#define load_vecdiff2(i)  \
	pow2((x1[i] - x2[i])/(x1[i]*x1[i]+y1[i]*y1[i]+z1[i]*z1[i])) + \
	pow2((y1[i] - y2[i])/(x1[i]*x1[i]+y1[i]*y1[i]+z1[i]*z1[i])) + \
	pow2((z1[i] - z2[i])/(x1[i]*x1[i]+y1[i]*y1[i]+z1[i]*z1[i]))   \

extern "C" __global__ void
reducerelmaxvecdiff2(float* __restrict__ x1, float* __restrict__ y1, float* __restrict__ z1,
                  float* __restrict__ x2, float* __restrict__ y2, float* __restrict__ z2,
                  float* __restrict__ dst, float initVal, int n) {
    reduce(load_vecdiff2, fmax, atomicFmaxabs)
}

